#include "hip/hip_runtime.h"
#include "/home/runner/work/slang/slang/prelude/slang-cuda-prelude.h"


#line 6679 "hlsl.meta.slang"
__device__ bool any_0(bool3  x_0)
{

#line 6679
    bool result_0 = false;

#line 6679
    int i_0 = int(0);

#line 6719
    for(;;)
    {

#line 6719
        if(i_0 < int(3))
        {
        }
        else
        {

#line 6719
            break;
        }

#line 6720
        if(result_0)
        {

#line 6720
            result_0 = true;

#line 6720
        }
        else
        {

#line 6720
            result_0 = (bool((_slang_vector_get_element(x_0, i_0))));

#line 6720
        }

#line 6719
        i_0 = i_0 + int(1);

#line 6719
    }

    return result_0;
}


#line 5 "test-texture-types.slang"
extern "C" __global__ void copyTexture(hipTextureObject_t srcTexture_0, hipSurfaceObject_t dstTexture_0)
{

    uint3  _S1 = uint3 {(blockIdx * blockDim + threadIdx).x, (blockIdx * blockDim + threadIdx).y, (blockIdx * blockDim + threadIdx).z};

    uint3  srcDims_0;
    {uint32_t w, h, d; asm("txq.width.b32 %0, [%3]; txq.height.b32 %1, [%3]; txq.depth.b32 %2, [%3];" : "=r"(w), "=r"(h), "=r"(d) : "l"((srcTexture_0))); *((&((&srcDims_0)->x))) = w;*((&((&srcDims_0)->y))) = h;*((&((&srcDims_0)->z))) = d;};
    uint3  dstDims_0;
    {uint32_t w, h, d; asm("txq.width.b32 %0, [%3]; txq.height.b32 %1, [%3]; txq.depth.b32 %2, [%3];" : "=r"(w), "=r"(h), "=r"(d) : "l"((srcTexture_0))); *((&((&dstDims_0)->x))) = w;*((&((&dstDims_0)->y))) = h;*((&((&dstDims_0)->z))) = d;};
    if(any_0(srcDims_0 != dstDims_0))
    {

#line 15
        return;
    }

#line 16
    if(any_0(_S1 >= dstDims_0))
    {

#line 17
        return;
    }

#line 18
    uint4  _S2 = make_uint4 (_S1.x, _S1.y, _S1.z, 0U);

#line 18
    int4  _S3 = make_int4 ((int)_S2.x, (int)_S2.y, (int)_S2.z, (int)_S2.w);

#line 18
    surf3Dwrite_convert<uint>(((tex3Dfetch_int<uint>((srcTexture_0), ((_S3)).x, ((_S3)).y, ((_S3)).z))), (dstTexture_0), ((_S1)).x * 1, ((_S1)).y, ((_S1)).z, SLANG_CUDA_BOUNDARY_MODE);
    return;
}

